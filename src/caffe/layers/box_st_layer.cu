#include "hip/hip_runtime.h"
#include <cfloat>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe
{

template <typename Dtype>
__global__ void BoxTransformForward(const int nthreads, const Dtype* bottom_theta_data, const Dtype* bottom_coordinate_data,
	Dtype* top_data)
{
	CUDA_KERNEL_LOOP(index, nthreads)
	{
		bottom_theta_data += index * 4;
		bottom_coordinate_data += index * 4;
		top_data[index * 4] = bottom_theta_data[0] * bottom_coordinate_data[0] + bottom_theta_data[2];
		top_data[index * 4 + 1] = bottom_theta_data[1] * bottom_coordinate_data[1] + bottom_theta_data[3];

		top_data[index * 4 + 2] = bottom_theta_data[0] * bottom_coordinate_data[2] + bottom_theta_data[2];
		top_data[index * 4 + 3] = bottom_theta_data[1] * bottom_coordinate_data[3] + bottom_theta_data[3];
	}
}

template <typename Dtype>
void BoxSpatialTransformerLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top)
{
	const Dtype* bottom_coordinate_data = bottom[0]->gpu_data();
	const Dtype* bottom_theta_data = bottom[1]->gpu_data();

	Dtype* top_data = top[0]->mutable_gpu_data();

	caffe_gpu_set(top[0]->count(), (Dtype)0, top_data);

	int batch_size = top[0]->shape(0);

	BoxTransformForward<Dtype><<<CAFFE_GET_BLOCKS(batch_size), CAFFE_CUDA_NUM_THREADS>>>(
		batch_size, bottom_theta_data, bottom_coordinate_data, top_data);
	CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void BoxTransformBackward(const int nthreads, const Dtype* top_diff, const Dtype* bottom_coordinate_data,
	Dtype* bottom_theta_diff)
{
	CUDA_KERNEL_LOOP(index, nthreads)
	{
		bottom_coordinate_data += index * 4;
		top_diff += index * 4;
		bottom_theta_diff[index * 4] = top_diff[0] * bottom_coordinate_data[0] + top_diff[2] * bottom_coordinate_data[2];
		bottom_theta_diff[index * 4 + 1] = top_diff[1] * bottom_coordinate_data[1] + top_diff[3] * bottom_coordinate_data[3];
		bottom_theta_diff[index * 4 + 2] = top_diff[0] + top_diff[2];
		bottom_theta_diff[index * 4 + 3] = top_diff[1] + top_diff[3];
	}
}

template <typename Dtype>
void BoxSpatialTransformerLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
	Dtype* bottom_theta_diff = bottom[1]->mutable_gpu_diff();
	const Dtype* top_diff = top[0]->gpu_diff();
	const Dtype* bottom_coordinate_data = bottom[0]->gpu_data();

	caffe_gpu_set(bottom[1]->count(), (Dtype)0, bottom_theta_diff);

	int batch_size = top[0]->shape(0);

	BoxTransformBackward<Dtype><<<CAFFE_GET_BLOCKS(batch_size), CAFFE_CUDA_NUM_THREADS>>>(
		batch_size, top_diff, bottom_coordinate_data, bottom_theta_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(BoxSpatialTransformerLayer);
}